#include <iostream>
#include <hip/hip_runtime.h>

#define N 16000


__global__ void matrixMulKernel(float* A, float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Column index

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k)
            sum += A[row * n + k] * B[k * n + col];
        C[row * n + col] = sum;
    }
}


int main() {
    size_t bytes = N * N * sizeof(float);

    // Allocate host memory
    float *h_A = new float[N * N];
    float *h_B = new float[N * N];
    float *h_C = new float[N * N];

    // Initialize input matrices
    // Seed the random number generator
    srand(time(NULL));
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Kernel launch parameters
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    // Launch kernel and note execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrixMulKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Output result
    std::cout << "Result matrix C:\n";
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j)
            std::cout << h_C[i * N + j] << " ";
        std::cout << "\n";
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}